#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hiprand.h>

#include "ranNumbers.h"

#define THS_MAX 256

#define FLAG \
        fprintf(stderr, "Flag in %s:%d\n", __FILE__, __LINE__);\

// Dormand-Prince coefficients 
#define A21 0.2
#define A31 0.075
#define A32 0.225
#define A41 (44.0/45.0)
#define A42 (-56.0/15.0)
#define A43 (32.0/9.0)
#define A51 (19372.0/6561.0)
#define A52 (-25360/2187.0)
#define A53 (64448.0/6561.0)
#define A54 (-212.0/729.0)
#define A61 (9017.0/3168.0)
#define A62 (-355.0/33.0)
#define A63 (46732.0/5247.0)
#define A64 (49.0/176.0)
#define A65 (-5103.0/18656.0)
#define A71 (35.0/384.0)
#define A73 (500.0/1113.0)
#define A74 (125.0/192.0)
#define A75 (-2187.0/6784.0)
#define A76 (11.0/84.0)

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- STRUCTURES =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

typedef struct 
{
	float X1;
	float X2;
	float X3;
	float X4;
} 
comp;

typedef struct 
{
	float min;
	float max;
} 
window;

typedef struct 
{
	float X1_0;
	float X2_0;
	float X3_0;
	float X4_0;

	float t0;
	float tN;
	float dt;

	float Vmin;
	float Tmax;

	int D;
	int Np;
	int nData;
	int nDataT;
	int qnData;
	int qFlag;
	int rssFlagT;
} 
param;

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- FUNCTIONS =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

long nextPow2(long x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

//-------------------------------------------------------------------------------

__device__ void modelInf(int idx, param pars, float *pop, comp Y, comp *dotY)
{
	int ii = 0;
	//float a0 = pow(10, pop[idx + ii]);
	ii++;
	float a1 = pow(10, pop[idx + ii]);
	ii++;
	float a2 = pow(10, pop[idx + ii]);
	ii++;
	float a3 = pow(10, pop[idx + ii]);
	ii++;
	float a4 = pow(10, pop[idx + ii]);
	ii++;
	float a5 = pow(10, pop[idx + ii]);
	ii++;
	float a6 = pow(10, pop[idx + ii]);

	/* Influenza model equations:
	  U = X1; I = X2; V = X3; T = X4;
	  a0 = V0 = X3_0
	  a1 = beta
	  a2 = del_I
	  a3 = p
	  a4 = c
	  a5 = r
	  a6 = del_T */

	float T0 = pars.X4_0;

	dotY->X1 = -a1*Y.X1*Y.X3; 
	dotY->X2 = a1*Y.X1*Y.X3 - a2*Y.X4*Y.X2;
	dotY->X3 = a3*Y.X2 - a4*Y.X3;
	dotY->X4 = a5*Y.X4*Y.X3 + a6*(T0 - Y.X4);

	return;
}

//-------------------------------------------------------------------------------

__device__ void deriv_step(int idx, param pars, float *pop, comp *Y)
{
	float h = pars.dt;
	comp Yold, Ytemp, k1, k2, k3, k4, k5, k6;

	// Old Y values
	Yold.X1 = Y->X1;
	Yold.X2 = Y->X2;
	Yold.X3 = Y->X3;
	Yold.X4 = Y->X4;
	
	modelInf(idx, pars, pop, Yold, &k1);

	Ytemp.X1 = Yold.X1 + h*A21*k1.X1;
	Ytemp.X2 = Yold.X2 + h*A21*k1.X2;
	Ytemp.X3 = Yold.X3 + h*A21*k1.X3;
	Ytemp.X4 = Yold.X4 + h*A21*k1.X4;

	modelInf(idx, pars, pop, Ytemp, &k2);

	Ytemp.X1 = Yold.X1 + h*(A31*k1.X1 + A32*k2.X1);
	Ytemp.X2 = Yold.X2 + h*(A31*k1.X2 + A32*k2.X2);
	Ytemp.X3 = Yold.X3 + h*(A31*k1.X3 + A32*k2.X3);
	Ytemp.X4 = Yold.X4 + h*(A31*k1.X4 + A32*k2.X4);

	modelInf(idx, pars, pop, Ytemp, &k3);

	Ytemp.X1 = Yold.X1 + h*(A41*k1.X1 + A42*k2.X1 + A43*k3.X1);
	Ytemp.X2 = Yold.X2 + h*(A41*k1.X2 + A42*k2.X2 + A43*k3.X2);
	Ytemp.X3 = Yold.X3 + h*(A41*k1.X3 + A42*k2.X3 + A43*k3.X3);
	Ytemp.X4 = Yold.X4 + h*(A41*k1.X4 + A42*k2.X4 + A43*k3.X4);

	modelInf(idx, pars, pop, Ytemp, &k4);

	Ytemp.X1 = Yold.X1 + h*(A51*k1.X1 + A52*k2.X1 + A53*k3.X1 + A54*k4.X1);
	Ytemp.X2 = Yold.X2 + h*(A51*k1.X2 + A52*k2.X2 + A53*k3.X2 + A54*k4.X2);
	Ytemp.X3 = Yold.X3 + h*(A51*k1.X3 + A52*k2.X3 + A53*k3.X3 + A54*k4.X3);
	Ytemp.X4 = Yold.X4 + h*(A51*k1.X4 + A52*k2.X4 + A53*k3.X4 + A54*k4.X4);

	modelInf(idx, pars, pop, Ytemp, &k5);

	Ytemp.X1 = Yold.X1 + h*(A61*k1.X1 + A62*k2.X1 + A63*k3.X1 + A64*k4.X1 + A65*k5.X1);
	Ytemp.X2 = Yold.X2 + h*(A61*k1.X2 + A62*k2.X2 + A63*k3.X2 + A64*k4.X2 + A65*k5.X2);
	Ytemp.X3 = Yold.X3 + h*(A61*k1.X3 + A62*k2.X3 + A63*k3.X3 + A64*k4.X3 + A65*k5.X3);
	Ytemp.X4 = Yold.X4 + h*(A61*k1.X4 + A62*k2.X4 + A63*k3.X4 + A64*k4.X4 + A65*k5.X4);

	modelInf(idx, pars, pop, Ytemp, &k6);

	// New Y values
	Y->X1 = Yold.X1 + h*(A71*k1.X1 + A73*k3.X1 + A74*k4.X1 + A75*k5.X1 + A76*k6.X1);
	Y->X2 = Yold.X2 + h*(A71*k1.X2 + A73*k3.X2 + A74*k4.X2 + A75*k5.X2 + A76*k6.X2);
	Y->X3 = Yold.X3 + h*(A71*k1.X3 + A73*k3.X3 + A74*k4.X3 + A75*k5.X3 + A76*k6.X3);
	Y->X4 = Yold.X4 + h*(A71*k1.X4 + A73*k3.X4 + A74*k4.X4 + A75*k5.X4 + A76*k6.X4);

	return;
}

//-------------------------------------------------------------------------------

__global__ void costFunction(param pars, float *pop, float *timeQt, float *dataQt,
		float *timeQt_T, float *dataQt_T, window *timeQl, window *dataQl,
		float *costFn)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= pars.Np) return;

	int penaltyFlag = 0;
	int rssFlag = 1;
	int rssFlagT = pars.rssFlagT;
	int qFlag = pars.qFlag;

	int nn = 0, nnT = 0, qnn = 0;
	int nData = pars.nData, nDataT = pars.nDataT, qnData = pars.qnData;
	float aux, sum2 = 0.0;
	float Vmin = pars.Vmin, Tmax = pars.Tmax;
	float tQt = timeQt[0];
	float tQt_T = timeQt_T[0];
	window tQl = timeQl[0];

	comp Y;
	int idx = ind*pars.D;
	float t = pars.t0;
	float h = pars.dt;

	// Initial values
	Y.X1 = pars.X1_0;
	Y.X2 = pars.X2_0;
	Y.X3 = pow(10, pop[idx]); // V0
	Y.X4 = pars.X4_0;

	while (t <= pars.tN)
	{
		// Dormand-Prince method to compute the next state
		deriv_step(idx, pars, pop, &Y);
		t += h;

		// Check for NaN and inf values
		if (isnan(Y.X1) || isnan(Y.X2) || isnan(Y.X3) || isnan(Y.X4)
			|| isinf(Y.X1) || isinf(Y.X2) || isinf(Y.X3) || isinf(Y.X4))
		{
			penaltyFlag = 1;
			break;
		}

		if (Y.X1 < 0.0) Y.X1 = 0.0;
		if (Y.X2 < 0.0) Y.X2 = 0.0;
		if (Y.X3 < 0.0) Y.X3 = 0.0;
		if (Y.X4 < 0.0) Y.X4 = 0.0;


		// This part calculates the quantitative RSS
		if (t >= tQt && rssFlag)
		{
			while (1)
			{
				// Data is already in log10
				if (Y.X3 == 0.0f) Y.X3 = 1e-38;
				aux = dataQt[nn] - log10(Y.X3); // Virus
				sum2 += aux*aux;
				nn++;

				if (nn >= nData)
				{
					rssFlag = 0;
					break;
				}

				if (timeQt[nn] != tQt)
				{
					tQt = timeQt[nn];
					break;
				}
			}
		}

		if (t >= tQt_T && rssFlagT)
		{
			while (1)
			{
				// Data is already in log10
				aux = dataQt_T[nnT] - log10(Y.X4); // T cells
				sum2 += aux*aux;
				nnT++;

				if (nnT >= nDataT)
				{
					rssFlagT = 0;
					break;
				}

				if (timeQt_T[nnT] != tQt_T)
				{
					tQt_T = timeQt_T[nnT];
					break;
				}
			}
		}

		// Qualitative penalties 
		if (t > tQl.min && qFlag == 1)
		{
			if (Y.X4 > dataQl[qnn].min && Y.X4 < dataQl[qnn].max) 
			{
				qnn++;
				if (qnn >= qnData) qFlag = 2;
				else tQl = timeQl[qnn];
			}
			else if (t > tQl.max)
			{
				penaltyFlag = 1;
				break;
			}
		}

		if (qFlag)
		{
			// Penalties for viral rebound
			if (!rssFlag && t>1.25*tQt && Y.X3 > Vmin)
			{
				penaltyFlag = 1;
				break;
			}

			// Penalties for reaching the maximum T cell level
			if (Y.X4 > Tmax)
			{
				penaltyFlag = 1;
				break;
			}
		}

		//if (!rssFlag && !rssFlagT && !qFlag) break;
	}

	if (isinf(sum2)) penaltyFlag = 1;
	costFn[ind] = penaltyFlag ? 1e38 : sum2;

	return;
}

//-------------------------------------------------------------------------------

__global__ void newPopulation(int Np, int D, float Cr, float Fm, float *randUni,
int3 *iiMut, float *lowerLim, float *upperLim, float *pop, float *newPop)
{
	int ind, jj, idx, auxInt, flag = 0;
	int3 iiM, idxM;
	float trial, auxL, auxU;

	ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= Np) return;

	iiM = iiMut[ind];

	for (jj=0; jj<D; jj++)
	{
		idx = ind*D + jj;

		auxL = lowerLim[jj];
		auxU = upperLim[jj];
		if (auxL == auxU)
		{
			newPop[idx] = auxL;
			continue;
		}

		idxM.x = iiM.x*D + jj;
		idxM.y = iiM.y*D + jj;
		idxM.z = iiM.z*D + jj;

		if (randUni[idx] <= Cr)
		{
			// DE/rand/1 || DE/best/1
			trial = pop[idxM.x] + Fm*(pop[idxM.y] - pop[idxM.z]); 
			// DE/current-to-best/1
			//trial = pop[idx] + Fm*(pop[idxM.x] - pop[idx])
			//		+ Fm*(pop[idxM.y] - pop[idxM.z]);

			if (trial < auxL) trial = auxL;
			if (trial > auxU) trial = auxU;

			newPop[idx] = trial;
			flag = 1;
		}
		else newPop[idx] = pop[idx];
	}

	// Ensure that at least one member of
	// the mutant vector is in the new population.
	if (!flag)
	{
		auxInt = ind*D;
		while (1)
		{
			jj = int(D*randUni[auxInt%(Np*D)]);
			if (jj == D) jj--;
			auxInt++;
			auxL = lowerLim[jj];
			auxU = upperLim[jj];
			if (auxL == auxU) continue;
			break;
		}

		idx = ind*D + jj;
		idxM.x = iiM.x*D + jj;
		idxM.y = iiM.y*D + jj;
		idxM.z = iiM.z*D + jj;

		// DE/rand/1 || DE/best/1
		trial = pop[idxM.x] + Fm*(pop[idxM.y] - pop[idxM.z]);
		// DE/current-to-best/1
		//trial = pop[idx] + Fm*(pop[idxM.x] - pop[idx])
		//		+ Fm*(pop[idxM.y] - pop[idxM.z]);

		if (trial < auxL) trial = auxL;
		if (trial > auxU) trial = auxU;

		newPop[idx] = trial;
	}

	return;
}

//-------------------------------------------------------------------------------

__global__ void selection(int Np, int D, float *pop, float *newPop,
float *costFn, float *newCostFn)
{
	int ind, jj, idx;

	ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= Np) return;

	if  (newCostFn[ind] > costFn[ind]) return;

	for (jj=0; jj<D; jj++)
	{
		idx = ind*D + jj;
		pop[idx] = newPop[idx];
	}
	costFn[ind] = newCostFn[ind];

	return;
}

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- MAIN =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

int main()
{
	/*+*+*+*+*+ FETCH DATA	+*+*+*+*+*/
	int nData, nn;
	float auxFloat;
	float *timeQt, *dataV_raw;
	char renglon[200], dirData[500], *linea;
	FILE *fileRead;

	// Read quantitative data
	sprintf(dirData, "viralLoad.csv");
	fileRead = fopen(dirData, "r");

	nData = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		nData++;
	}
	fclose(fileRead);

	if (nData == 0)
	{
		printf("Error: Empty file in %s\n", dirData);
		exit (1);
	}
	nData--;

	hipMallocManaged(&timeQt, nData*sizeof(float));
	dataV_raw = (float *) malloc(nData*sizeof(float));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQt[nn] = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataV_raw[nn] = log10(auxFloat);

		nn++;
	}
	fclose(fileRead);

	// Read quantitative CD8 T cell data
	int nDataT;
	float *timeQt_T, *dataT_raw;

	sprintf(dirData, "CD8_data.csv");
	fileRead = fopen(dirData, "r");

	nDataT = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		nDataT++;
	}
	fclose(fileRead);

	if (nDataT == 0)
	{
		printf("Error: Empty file in %s\n", dirData);
		exit (1);
	}
	nDataT--;

	hipMallocManaged(&timeQt_T, nDataT*sizeof(float));
	dataT_raw = (float *) malloc(nDataT*sizeof(float));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQt_T[nn] = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataT_raw[nn] = log10(auxFloat);

		nn++;
	}
	fclose(fileRead);

	// Read qualitative data
	int qnData;
	window *timeQl, *dataQl;

	sprintf(dirData, "qualTcell.csv");
	fileRead = fopen(dirData, "r");

	qnData = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		qnData++;
	}
	fclose(fileRead);

	if (qnData == 0)
	{
		printf("Error in qualitative data\n");
		exit (1);
	}
	qnData--;

	hipMallocManaged(&timeQl, qnData*sizeof(window));
	hipMallocManaged(&dataQl, qnData*sizeof(window));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQl[nn].min = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQl[nn].max = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQl[nn].min = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQl[nn].max = auxFloat;

		nn++;
	}
	fclose(fileRead);

	// Scale qualitative windows for T cell level in influenza model
	float T0 = 1e6, Tmax = 1e7;
	for (nn=0; nn<qnData; nn++)
	{
		auxFloat = dataQl[nn].min;
		dataQl[nn].min = auxFloat*(Tmax-T0) + T0;
		auxFloat = dataQl[nn].max;
		dataQl[nn].max = auxFloat*(Tmax-T0) + T0;
	}

    	/*+*+*+*+*+ FETCH PARAMETERS +*+*+*+*+*/
	int Np, itMax, seed, D, bootFlag, rssFlagT, qFlag;
	float Fm, Cr, t0, tN, dt;
	int err_flag = 0;

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* DE parameters */
	// Population of parameter vector
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &Np);

	// Maximum iterations
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &itMax);

	// Recombination probability
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &Cr);

	// Mutation factor
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &Fm);

	// Seed for random numbers
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &seed);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* Initial conditions for ODE solve */
	// Initial time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &t0);

	// Final time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &tN);

	// Step time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &dt);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* Parameters to estimate */
	// Number of parameters to estimate
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &D);

	// Activate sampling for Bootstraping?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &bootFlag);

	// Include qualitative fit?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &qFlag);

	// Test raw T cell data?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &rssFlagT);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	if (err_flag)
	{
		printf("Error: Something is wrong in the parameter file (.param)\n");
		exit (1);
	}

	param pars;

	pars.D = D;
	pars.t0 = t0;
	pars.tN = tN;
	pars.Np = Np;
	pars.dt = dt;
	pars.nData = nData;
	pars.nDataT = nDataT;
	pars.qnData = qnData;
	pars.qFlag = qFlag;
	if (!qFlag) pars.rssFlagT = rssFlagT;
	else pars.rssFlagT = 0;

	pars.Vmin = 50.0f; // Minimum threshold of viral load
	pars.Tmax = Tmax;

	// Initial values
        pars.X1_0 = 1e7;
        pars.X2_0 = 0.0;
        pars.X3_0 = 0.0;
        pars.X4_0 = T0;

	float *lowerLim, *upperLim, *pop;
	int ii, jj, idx;

	hipMallocManaged(&lowerLim, D*sizeof(float));
	hipMallocManaged(&upperLim, D*sizeof(float));

	float aux;
	float auxL, auxU;

	for (jj=0; jj<D; jj++)
	{
		if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
		else sscanf(renglon, "[%f : %f]", &auxL, &auxU);
		lowerLim[jj] = auxL;
		upperLim[jj] = auxU;
		//lowerLim[jj] = log10(auxL);
		//upperLim[jj] = log10(auxU);

		if (auxL > auxU)
		{
			printf("Error: Invalid range in parameter %d (param file)\n", jj);
			exit (1);
		}
	}

	hipMallocManaged(&pop, Np*D*sizeof(float));

	// Initialize random numbers
	if (seed < 0) seed *= -1;
	Ran ranUni(seed);

	// Initialize population
	for (jj=0; jj<D; jj++)
	{
		aux = upperLim[jj] - lowerLim[jj];
		for (ii=0; ii<Np; ii++)
		{
			idx = ii*D + jj;
			if (aux == 0.0) pop[idx] = lowerLim[jj];
			else pop[idx] = lowerLim[jj] + aux*ranUni.doub();
		}
	}

	// Sampling for bootstrap
	float *dataQt;
	hipMallocManaged(&dataQt, nData*sizeof(float));

	float *dataQt_T;
	hipMallocManaged(&dataQt_T, nDataT*sizeof(float));

	if (bootFlag)
	{
		int auxInt;
		int tt;
		float oldTime;

		tt = 0;
		oldTime = timeQt[0];
		for (nn=1; nn<=nData; nn++)
		{
			tt++;

			if(nn < nData) if (oldTime == timeQt[nn]) continue;

			for (ii=nn-tt; ii<nn; ii++)
			{
				auxInt = tt*ranUni.doub();
				// Using modulo for safety (auxInt!=tt)
				dataQt[ii] = dataV_raw[nn-tt+(auxInt%tt)];
			}

			if (nn == nData) break;

			tt = 0;
			oldTime = timeQt[nn];
		}

		tt = 0;
		oldTime = timeQt_T[0];
		for (nn=1; nn<=nDataT; nn++)
		{
			tt++;

			if(nn < nDataT) if (oldTime == timeQt_T[nn]) continue;

			for (ii=nn-tt; ii<nn; ii++)
			{
				auxInt = tt*ranUni.doub();
				// Using modulo for safety (auxInt!=tt)
				dataQt_T[ii] = dataT_raw[nn-tt+(auxInt%tt)];
			}

			if (nn == nDataT) break;

			tt = 0;
			oldTime = timeQt_T[nn];
		}
	}
	else
	{
		for (nn=0; nn<nData; nn++) dataQt[nn] = dataV_raw[nn];
		for (nn=0; nn<nDataT; nn++) dataQt_T[nn] = dataT_raw[nn];
	}
	free(dataV_raw);
	free(dataT_raw);

	int ths, blks;
	float *costFn, *d_newCostFn;

	hipMallocManaged(&costFn, Np*sizeof(float));
	hipMalloc(&d_newCostFn, Np*sizeof(float));

	// Estimate the number of threads and blocks for the GPU
	ths = (Np < THS_MAX) ? nextPow2(Np) : THS_MAX;
	blks = 1 + (Np - 1)/ths;

	// Calculate cost function values
	costFunction<<<blks, ths>>>(pars, pop, timeQt, dataQt, timeQt_T, dataQt_T,
			timeQl, dataQl, costFn);
	hipDeviceSynchronize();


    	/*+*+*+*+*+ START OPTIMIZATION +*+*+*+*+*/
	int it, xx, yy, zz;
	int3 *iiMut;
	float *d_randUni, *d_newPop;
	float minVal;
	int iiMin;
	hiprandGenerator_t gen;

	hipMallocManaged(&iiMut, Np*sizeof(int3));
	hipMalloc(&d_newPop, Np*D*sizeof(float));

	// Initialize random numbers with a standard normal distribution
	// I use cuRand libraries 
	hipMalloc(&d_randUni, Np*D*sizeof(float)); // Array only for GPU
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);


	// Start iterations
	for (it=0; it<itMax; it++)
	{
		//minVal = costFn[0];
		//iiMin = 0;
		//for(ii=1; ii<Np; ii++) if (minVal > costFn[ii])
		//{
		//	minVal = costFn[ii];
		//	iiMin = ii;
		//}

		//xx = iiMin; // best
		for (ii=0; ii<Np; ii++)
		{
			do xx = Np*ranUni.doub(); while (xx == ii || xx == Np);
			do yy = Np*ranUni.doub(); while (yy == ii || yy == xx || yy == Np);
			do zz = Np*ranUni.doub(); while (zz == ii || zz == yy || zz == xx || zz == Np);

			iiMut[ii].x = xx; iiMut[ii].y = yy; iiMut[ii].z = zz;
		}

		// Generate random numbers and then update positions
		hiprandGenerateUniform(gen, d_randUni, Np*D);

		// Generate new population
		newPopulation<<<blks, ths>>>(Np, D, Cr, Fm, d_randUni, iiMut, lowerLim, upperLim, pop, d_newPop);

		// Calculate cost function values
		costFunction<<<blks, ths>>>(pars, d_newPop, timeQt, dataQt, timeQt_T, dataQt_T, timeQl, dataQl, d_newCostFn);

		// Select the best vectors between new ones and old ones
		selection<<<blks, ths>>>(Np, D, pop, d_newPop, costFn, d_newCostFn);

		hipDeviceSynchronize();

		// Save population for analysis
		//if (!flag) for (ii=0; ii<Np; ii++)
		//{
		//	//if (costFn[ii] == 1e10) continue;
		//	for(jj=0; jj<D; jj++) fprintf(fPars, "%.3e ", pop[ii*D + jj]);
		//	fprintf(fPars, "%.3e %d\n", costFn[ii], it);
		//}
	}

	//fclose(fPars);

	// Minimum of the population
	minVal = costFn[0];
	iiMin = 0;
	for (ii=1; ii<Np; ii++) if (minVal > costFn[ii])
	{
		minVal = costFn[ii];
		iiMin = ii;
	}

	FILE *fBestPars;
	fBestPars = fopen("bestPars.dat", "a");
	//for (jj=0; jj<D; jj++) fprintf(fBestPars, "%.4e ", pow(10, pop[iiMin*D + jj]));
	for (jj=0; jj<D; jj++) fprintf(fBestPars, "%.4f ", pop[iiMin*D + jj]);
	fprintf(fBestPars, "%.4e\n", minVal);
	fclose(fBestPars);

	printf("FINISHED\n");

	hipFree(timeQt);
	hipFree(timeQt_T);
	hipFree(timeQl);
	hipFree(lowerLim);
	hipFree(upperLim);
	hipFree(dataQt);
	hipFree(dataQt_T);
	hipFree(dataQl);
	hipFree(iiMut);
	hipFree(pop);
	hipFree(d_newPop);
	hipFree(costFn);
	hipFree(d_newCostFn);
	hipFree(d_randUni);
	hiprandDestroyGenerator(gen);

	exit(0);
}
