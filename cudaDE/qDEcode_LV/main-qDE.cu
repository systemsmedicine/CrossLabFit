#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hiprand.h>

#include "ranNumbers.h"

#define THS_MAX 256

#define FLAG \
        fprintf(stderr, "Flag in %s:%d\n", __FILE__, __LINE__);\

// Dormand-Prince coefficients
#define A21 0.2
#define A31 0.075
#define A32 0.225
#define A41 (44.0/45.0)
#define A42 (-56.0/15.0)
#define A43 (32.0/9.0)
#define A51 (19372.0/6561.0)
#define A52 (-25360/2187.0)
#define A53 (64448.0/6561.0)
#define A54 (-212.0/729.0)
#define A61 (9017.0/3168.0)
#define A62 (-355.0/33.0)
#define A63 (46732.0/5247.0)
#define A64 (49.0/176.0)
#define A65 (-5103.0/18656.0)
#define A71 (35.0/384.0)
#define A73 (500.0/1113.0)
#define A74 (125.0/192.0)
#define A75 (-2187.0/6784.0)
#define A76 (11.0/84.0)

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- STRUCTURES =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

typedef struct 
{
	float X1;
	float X2;
	float X3;
} 
comp;

typedef struct
{
        float min;
        float max;
}
window;

typedef struct 
{
	float X1_0;
	float X2_0;
	float X3_0;

	float a1;
	float a2;
	float a3;
	float a4;
	float a5;
	float a6;
	float a7;

	float t0;
	float tN;
	float dt;

	int D;
	int Np;
	int nData_X1;
	int nData_X3;
	int qnData_X3;
	int rssFlag_X3;
	int qFlag_X3;
} 
param;

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- FUNCTIONS =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

// Encuentra la siguiente potencia de dos
long nextPow2(long x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

//-------------------------------------------------------------------------------

__device__ void modelLV(int idx, param pars, float *pop, comp Y, comp *dotY)
{
	int ii = 0;
	float a0 = pop[idx + ii];
	ii++;
	float a1 = pop[idx + ii];
	ii++;
	float a2 = pop[idx + ii];
	ii++;
	float a3 = pop[idx + ii];
	ii++;
	float a4 = pop[idx + ii];
	ii++;
	float a5 = pop[idx + ii];
	ii++;
	float a6 = pop[idx + ii];
	ii++;
	float a7 = pop[idx + ii];
	ii++;
	float a8 = pop[idx + ii];
	ii++;
	float a9 = pop[idx + ii];

	// Three-species LV model
	dotY->X1 = a0*Y.X1 - a1*Y.X1 - a2*Y.X1*Y.X2 + a3*Y.X1*Y.X3;
	dotY->X2 = a4*Y.X1*Y.X2 - a5*Y.X2 - a6*Y.X2*Y.X3;
	dotY->X3 = -a7*Y.X1*Y.X3 + a8*Y.X2*Y.X3 - a9*Y.X3;

	return;
}

//-------------------------------------------------------------------------------

__device__ void deriv_step(int idx, param pars, float *pop, comp *Y)
{
        float h = pars.dt;
        comp Yold, Ytemp, k1, k2, k3, k4, k5, k6;

        // Old Y values
        Yold.X1 = Y->X1;
        Yold.X2 = Y->X2;
        Yold.X3 = Y->X3;

        modelLV(idx, pars, pop, Yold, &k1);

        Ytemp.X1 = Yold.X1 + h*A21*k1.X1;
        Ytemp.X2 = Yold.X2 + h*A21*k1.X2;
        Ytemp.X3 = Yold.X3 + h*A21*k1.X3;

        modelLV(idx, pars, pop, Ytemp, &k2);

        Ytemp.X1 = Yold.X1 + h*(A31*k1.X1 + A32*k2.X1);
        Ytemp.X2 = Yold.X2 + h*(A31*k1.X2 + A32*k2.X2);
        Ytemp.X3 = Yold.X3 + h*(A31*k1.X3 + A32*k2.X3);

        modelLV(idx, pars, pop, Ytemp, &k3);

        Ytemp.X1 = Yold.X1 + h*(A41*k1.X1 + A42*k2.X1 + A43*k3.X1);
        Ytemp.X2 = Yold.X2 + h*(A41*k1.X2 + A42*k2.X2 + A43*k3.X2);
        Ytemp.X3 = Yold.X3 + h*(A41*k1.X3 + A42*k2.X3 + A43*k3.X3);

        modelLV(idx, pars, pop, Ytemp, &k4);

        Ytemp.X1 = Yold.X1 + h*(A51*k1.X1 + A52*k2.X1 + A53*k3.X1 + A54*k4.X1);
        Ytemp.X2 = Yold.X2 + h*(A51*k1.X2 + A52*k2.X2 + A53*k3.X2 + A54*k4.X2);
        Ytemp.X3 = Yold.X3 + h*(A51*k1.X3 + A52*k2.X3 + A53*k3.X3 + A54*k4.X3);

        modelLV(idx, pars, pop, Ytemp, &k5);

        Ytemp.X1 = Yold.X1 + h*(A61*k1.X1 + A62*k2.X1 + A63*k3.X1 + A64*k4.X1 + A65*k5.X1);
        Ytemp.X2 = Yold.X2 + h*(A61*k1.X2 + A62*k2.X2 + A63*k3.X2 + A64*k4.X2 + A65*k5.X2);
        Ytemp.X3 = Yold.X3 + h*(A61*k1.X3 + A62*k2.X3 + A63*k3.X3 + A64*k4.X3 + A65*k5.X3);

        modelLV(idx, pars, pop, Ytemp, &k6);

        // New Y values
        Y->X1 = Yold.X1 + h*(A71*k1.X1 + A73*k3.X1 + A74*k4.X1 + A75*k5.X1 + A76*k6.X1);
        Y->X2 = Yold.X2 + h*(A71*k1.X2 + A73*k3.X2 + A74*k4.X2 + A75*k5.X2 + A76*k6.X2);
        Y->X3 = Yold.X3 + h*(A71*k1.X3 + A73*k3.X3 + A74*k4.X3 + A75*k5.X3 + A76*k6.X3);

        return;
}

//-------------------------------------------------------------------------------

__global__ void costFunction(param pars, float *pop, float *timeQt_X1, float *dataQt_X1,
		float *timeQt_X3, float *dataQt_X3, window *timeQl_X3, window *dataQl_X3,
		float *costFn)
{
	int ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= pars.Np) return;

	int penaltyFlag = 0;
	int rssFlag_X1 = 1;
	int rssFlag_X3 = pars.rssFlag_X3;
	int qFlag_X3 = pars.qFlag_X3;

	int nn_X1 = 0, nn_X3 = 0, qnn_X3 = 0;
        int nData_X1 = pars.nData_X1,
	    nData_X3 = pars.nData_X3,
	    qnData_X3 = pars.qnData_X3;
        float aux, sum2 = 0.0;
        float tQt_X1 = timeQt_X1[0];
        float tQt_X3 = timeQt_X3[0];
        window tQl_X3 = timeQl_X3[0];

	comp Y;
	int idx = ind*pars.D;
	float t = pars.t0;
	float h = pars.dt;

	// Initial values
	Y.X1 = pars.X1_0;
	Y.X2 = pars.X2_0;
	Y.X3 = pars.X3_0;

	while (t <= pars.tN)
	{
		// Dormand-Prince method to compute the next state
                deriv_step(idx, pars, pop, &Y);
                t += h;

		// Check for NaN and inf values
                if (isnan(Y.X1) || isnan(Y.X2) || isnan(Y.X3)
                        || isinf(Y.X1) || isinf(Y.X2) || isinf(Y.X3))
                {
                        penaltyFlag = 1;
                        break;
                }

		if (Y.X1 < 0.0) Y.X1 = 0.0;
                if (Y.X2 < 0.0) Y.X2 = 0.0;
                if (Y.X3 < 0.0) Y.X3 = 0.0;

		// This part calculates the quantitative RSS
		if (t >= tQt_X1 && rssFlag_X1)
		{
			while (1)
			{
				aux = dataQt_X1[nn_X1] - Y.X1;
				sum2 += aux*aux;
				nn_X1++;

				if (nn_X1 >= nData_X1)
				{
					rssFlag_X1 = 0;
					break;
				}

				if (timeQt_X1[nn_X1] != tQt_X1)
				{
					tQt_X1 = timeQt_X1[nn_X1];
					break;

				}
			}
		}

		if (t >= tQt_X3 && rssFlag_X3)
		{
			while (1)
			{
				aux = dataQt_X3[nn_X3] - Y.X3;
				sum2 += aux*aux;
				nn_X3++;

				if (nn_X3 >= nData_X3)
				{
					rssFlag_X3 = 0;
					break;
				}

				if (timeQt_X3[nn_X3] != tQt_X3)
				{
					tQt_X1 = timeQt_X1[nn_X3];
					break;

				}
			}
		}

		// Qualitative penalties
		if (t > tQl_X3.min && qFlag_X3)
		{
			if (Y.X3 > dataQl_X3[qnn_X3].min && Y.X3 < dataQl_X3[qnn_X3].max)
                        {
                                qnn_X3++;
                                if (qnn_X3 >= qnData_X3) qFlag_X3 = 0;
                                else tQl_X3 = timeQl_X3[qnn_X3];
                        }
                        else if (t > tQl_X3.max)
                        {
                                penaltyFlag = 1;
                                break;
                        }
		}

		if (!rssFlag_X1 && !rssFlag_X3 && !qFlag_X3) break;
	}

	if (isinf(sum2)) penaltyFlag = 1;
        costFn[ind] = penaltyFlag ? 1e38 : sum2;

	return;
}

//-------------------------------------------------------------------------------

__global__ void newPopulation(int Np, int D, float Cr, float Fm, float *randUni,
		int3 *iiMut, float *lowerLim, float *upperLim, float *pop, float *newPop)
{
	int ind, jj, idx, auxInt, flag = 0;
	int3 iiM, idxM;
	float trial, auxL, auxU;

	ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= Np) return;

	iiM = iiMut[ind];

	for (jj=0; jj<D; jj++)
	{
		idx = ind*D + jj;

		auxL = lowerLim[jj];
		auxU = upperLim[jj];
		if (auxL == auxU)
		{
			newPop[idx] = auxL;
			continue;
		}

		idxM.x = iiM.x*D + jj;
		idxM.y = iiM.y*D + jj;
		idxM.z = iiM.z*D + jj;

		if (randUni[idx] <= Cr)
		{
			// DE/rand/1 || DE/best/1
                        trial = pop[idxM.x] + Fm*(pop[idxM.y] - pop[idxM.z]);
                        // DE/current-to-best/1
                        //trial = pop[idx] + Fm*(pop[idxM.x] - pop[idx])
                        //              + Fm*(pop[idxM.y] - pop[idxM.z]);

			if (trial < auxL) trial = auxL;
			if (trial > auxU) trial = auxU;

			newPop[idx] = trial;
			flag = 1;
		}
		else newPop[idx] = pop[idx];
	}

	// Se asegura que exista al menos un elemento
	// del vector mutante en la nueva población
	if (!flag)
	{
		auxInt = ind*D;
		while (1)
		{
			jj = int(D*randUni[auxInt%(Np*D)]);
			if (jj == D) jj--;
			auxInt++;
			auxL = lowerLim[jj];
			auxU = upperLim[jj];
			if (auxL == auxU) continue;
			break;
		}

		idx = ind*D + jj;
		idxM.x = iiM.x*D + jj;
		idxM.y = iiM.y*D + jj;
		idxM.z = iiM.z*D + jj;

		// DE/rand/1 || DE/best/1
                trial = pop[idxM.x] + Fm*(pop[idxM.y] - pop[idxM.z]);
                // DE/current-to-best/1
                //trial = pop[idx] + Fm*(pop[idxM.x] - pop[idx])
                //              + Fm*(pop[idxM.y] - pop[idxM.z]);

		if (trial < auxL) trial = auxL;
		if (trial > auxU) trial = auxU;

		newPop[idx] = trial;
	}

	return;
}

//-------------------------------------------------------------------------------

__global__ void selection(int Np, int D, float *pop, float *newPop,
		float *costFn, float *newCostFn)
{
	int ind, jj, idx;

	ind = threadIdx.x + blockIdx.x*blockDim.x;
	if (ind >= Np) return;

	if (newCostFn[ind] > costFn[ind]) return;

	for (jj=0; jj<D; jj++)
	{
		idx = ind*D + jj;
		pop[idx] = newPop[idx];
	}
	costFn[ind] = newCostFn[ind];

	return;
}

/*=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=- MAIN =-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-=-*/

int main()
{
	/*+*+*+*+*+ FETCH DATA	+*+*+*+*+*/
	int nData_X1, nData_X3, nn;
	float auxFloat;
	float *timeQt_X1, *timeQt_X3;
	float *dataQt_X1_raw, *dataQt_X3_raw;
	char renglon[200], dirData[500], *linea;
	FILE *fileRead;

	// Raw data for X1 variable
	sprintf(dirData, "LVdata_noise.csv");
	fileRead = fopen(dirData, "r");

	nData_X1 = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		nData_X1++;
	}
	fclose(fileRead);

	if (nData_X1 == 0)
	{
		printf("Error: Empty file in %s\n", dirData);
		exit (1);
	}
	nData_X1--;

	hipMallocManaged(&timeQt_X1, nData_X1*sizeof(float));
	dataQt_X1_raw = (float *) malloc(nData_X1*sizeof(float));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQt_X1[nn] = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQt_X1_raw[nn] = auxFloat;

		nn++;
	}
	fclose(fileRead);

	// Raw data for X3 variable
	sprintf(dirData, "LVdata_noise_X3.csv");
	fileRead = fopen(dirData, "r");

	nData_X3 = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		nData_X3++;
	}
	fclose(fileRead);

	if (nData_X3 == 0)
	{
		printf("Error: Empty file in %s\n", dirData);
		exit (1);
	}
	nData_X3--;

	hipMallocManaged(&timeQt_X3, nData_X3*sizeof(float));
	dataQt_X3_raw = (float *) malloc(nData_X3*sizeof(float));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQt_X3[nn] = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQt_X3_raw[nn] = auxFloat;

		nn++;
	}
	fclose(fileRead);

	// Read qualitative data for X3 variable
	int qnData_X3;
	window *timeQl_X3, *dataQl_X3;

	sprintf(dirData, "qualX3cell.csv");
	fileRead = fopen(dirData, "r");

	qnData_X3 = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;
		qnData_X3++;
	}
	fclose(fileRead);

	if (qnData_X3 == 0)
	{
		printf("Error: Empty file in %s\n", dirData);
		exit (1);
	}
	qnData_X3--;

	hipMallocManaged(&timeQl_X3, qnData_X3*sizeof(window));
	hipMallocManaged(&dataQl_X3, qnData_X3*sizeof(window));

	fileRead = fopen(dirData, "r");
	if (fgets(renglon, sizeof(renglon), fileRead) == NULL) exit (1);

	nn = 0;
	while (1)
	{
		if (fgets(renglon, sizeof(renglon), fileRead) == NULL) break;

		linea = strtok(renglon, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQl_X3[nn].min = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		timeQl_X3[nn].max = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQl_X3[nn].min = auxFloat;

		linea = strtok(NULL, ",");
		sscanf(linea, "%f", &auxFloat);
		dataQl_X3[nn].max = auxFloat;

		nn++;
	}
	fclose(fileRead);

    	/*+*+*+*+*+ FETCH PARAMETERS +*+*+*+*+*/
	int Np, itMax, seed, D, bootFlag, rssFlag_X3, qFlag_X3;
	float Fm, Cr, t0, tN, dt;
	int err_flag = 0;

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* DE parameters */
	// Population of parameter vector
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &Np);

	// Maximum iterations
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &itMax);

	// Recombination probability
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &Cr);

	// Mutation factor
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &Fm);

	// Seed for random numbers
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &seed);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* Initial conditions for ODE solve */
	// Initial time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &t0);

	// Final time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &tN);

	// Step time
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%f", &dt);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	/* Parameters to estimate */
	// Number of parameters to estimate
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &D);

	// Activate sampling for Bootstraping?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &bootFlag);

	// Include qualitative fit?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &qFlag_X3);

	// Include raw X3 data?
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	else sscanf(renglon, "%d", &rssFlag_X3);

	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
	if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;

	if (err_flag)
	{
		printf("Error: Something is wrong in the parameter file (.param)\n");
		exit (1);
	}

	param pars;

	pars.D = D;
	pars.t0 = t0;
	pars.tN = tN;
	pars.Np = Np;
	pars.dt = dt;
	pars.nData_X1 = nData_X1;
	pars.nData_X3 = nData_X3;
	pars.qnData_X3 = qnData_X3;
	pars.qFlag_X3 = qFlag_X3;
	if (!qFlag_X3) pars.rssFlag_X3 = rssFlag_X3;
        else pars.rssFlag_X3 = 0;

	// Initial values
        pars.X1_0 = 4.0;
        pars.X2_0 = 2.0;
        pars.X3_0 = 1.0;

	float *lowerLim, *upperLim, *pop;
	int ii, jj, idx;

	hipMallocManaged(&lowerLim, D*sizeof(float));
	hipMallocManaged(&upperLim, D*sizeof(float));

	float aux;
	float auxL, auxU;

	for (jj=0; jj<D; jj++)
	{
		if (fgets(renglon, sizeof(renglon), stdin) == NULL) err_flag = 1;
		else sscanf(renglon, "[%f : %f]", &auxL, &auxU);
		lowerLim[jj] = auxL;
		upperLim[jj] = auxU;

		if (auxL > auxU)
		{
			printf("Error: Invalid range in parameter %d (param file)\n", jj);
			exit (1);
		}
	}

	hipMallocManaged(&pop, Np*D*sizeof(float));

	// Initialize random numbers
	if (seed < 0) seed *= -1;
	Ran ranUni(seed);

	// Initialize population
	for (jj=0; jj<D; jj++)
	{
		aux = upperLim[jj] - lowerLim[jj];
		for (ii=0; ii<Np; ii++)
		{
			idx = ii*D + jj;
			if (aux == 0.0) pop[idx] = lowerLim[jj];
			else pop[idx] = lowerLim[jj] + aux*ranUni.doub();
		}
	}

	// Sampling for bootstrap
	float *dataQt_X1;
	hipMallocManaged(&dataQt_X1, nData_X1*sizeof(float));

	float *dataQt_X3;
	hipMallocManaged(&dataQt_X3, nData_X3*sizeof(float));

	if (bootFlag)
	{
		int auxInt;
		int tt;
		float oldTime;

		tt = 0;
		oldTime = timeQt_X1[0];
		for (nn=1; nn<=nData_X1; nn++)
		{
			tt++;

			if (nn < nData_X1) if (oldTime == timeQt_X1[nn]) continue;

			for (ii=nn-tt; ii<nn; ii++)
			{
				auxInt = tt*ranUni.doub();
				// Using modulo for safety (auxInt!=tt)
				dataQt_X1[ii] = dataQt_X1_raw[nn-tt+(auxInt%tt)];
			}

			if (nn == nData_X1) break;
			
			tt = 0;
			oldTime = timeQt_X1[nn];
		}

		tt = 0;
		oldTime = timeQt_X3[0];
		for (nn=1; nn<=nData_X3; nn++)
		{
			tt++;

			if (nn < nData_X3) if (oldTime == timeQt_X3[nn]) continue;

			for (ii=nn-tt; ii<nn; ii++)
			{
				auxInt = tt*ranUni.doub();
				// Using modulo for safety (auxInt!=tt)
				dataQt_X3[ii] = dataQt_X3_raw[nn-tt+(auxInt%tt)];
			}

			if (nn == nData_X3) break;
			
			tt = 0;
			oldTime = timeQt_X3[nn];
		}
	}
	else
	{
		for (nn=0; nn<nData_X1; nn++) dataQt_X1[nn] = dataQt_X1_raw[nn];
		for (nn=0; nn<nData_X3; nn++) dataQt_X3[nn] = dataQt_X3_raw[nn];
	}
	free(dataQt_X1_raw);
	free(dataQt_X3_raw);

	int ths, blks;
	float *costFn, *d_newCostFn;

	hipMallocManaged(&costFn, Np*sizeof(float));
	hipMalloc(&d_newCostFn, Np*sizeof(float));

	// Estimate the number of threads and blocks for the GPU
	ths = (Np < THS_MAX) ? nextPow2(Np) : THS_MAX;
	blks = 1 + (Np - 1)/ths;

	// Calcula el valor de la función objetivo
	costFunction<<<blks, ths>>>(pars, pop, timeQt_X1, dataQt_X1, timeQt_X3, dataQt_X3,
			timeQl_X3, dataQl_X3, costFn);
	hipDeviceSynchronize();

    	/*+*+*+*+*+ START OPTIMIZATION +*+*+*+*+*/
	int it, xx, yy, zz;
	int3 *iiMut;
	float *d_randUni, *d_newPop;
	float minVal;
	int iiMin;
	hiprandGenerator_t gen;

	hipMallocManaged(&iiMut, Np*sizeof(int3));
	hipMalloc(&d_newPop, Np*D*sizeof(float));

	// Initialize random numbers with a standard normal distribution
	// I use cuRand libraries 
	hipMalloc(&d_randUni, Np*D*sizeof(float)); // Array only for GPU
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);

	// Start iterations
	for (it=0; it<itMax; it++)
	{
		//minVal = costFn[0];
		//iiMin = 0;
		//for(ii=1; ii<Np; ii++) if (minVal > costFn[ii])
		//{
		//	minVal = costFn[ii];
		//	iiMin = ii;
		//}

		//xx = iiMin; // best
		for (ii=0; ii<Np; ii++)
		{
			do xx = Np*ranUni.doub(); while (xx == ii || xx == Np);
			do yy = Np*ranUni.doub(); while (yy == ii || yy == xx || yy == Np);
			do zz = Np*ranUni.doub(); while (zz == ii || zz == yy || zz == xx || zz == Np);

			iiMut[ii].x = xx; iiMut[ii].y = yy; iiMut[ii].z = zz;
		}

		// Generate random numbers and then update positions
		hiprandGenerateUniform(gen, d_randUni, Np*D);

		// Generate new population
		newPopulation<<<blks, ths>>>(Np, D, Cr, Fm, d_randUni, iiMut,
				lowerLim, upperLim, pop, d_newPop);

		// Calculate cost function values
		costFunction<<<blks, ths>>>(pars, d_newPop, timeQt_X1, dataQt_X1, timeQt_X3, dataQt_X3,
				timeQl_X3, dataQl_X3, d_newCostFn);

		// Select the best vectors between new ones and old ones
		selection<<<blks, ths>>>(Np, D, pop, d_newPop, costFn, d_newCostFn);

		hipDeviceSynchronize();
	}


	// Minimum of the population
	minVal = costFn[0];
	iiMin = 0;
	for (ii=1; ii<Np; ii++) if (minVal > costFn[ii])
	{
		minVal = costFn[ii];
		iiMin = ii;
	}

	FILE *fBestPars;
	fBestPars = fopen("bestPars.dat", "a");
	for (jj=0; jj<D; jj++) fprintf(fBestPars, "%.4e ", pop[iiMin*D + jj]);
	fprintf(fBestPars, "%.4e\n", minVal);
	fclose(fBestPars);

	printf("FINISHED\n");

	hipFree(timeQt_X1);
	hipFree(timeQt_X3);
	hipFree(timeQl_X3);
	hipFree(lowerLim);
	hipFree(upperLim);
	hipFree(dataQt_X1);
	hipFree(dataQt_X3);
	hipFree(dataQl_X3);
	hipFree(iiMut);
	hipFree(pop);
	hipFree(d_newPop);
	hipFree(costFn);
	hipFree(d_newCostFn);
	hipFree(d_randUni);
	hiprandDestroyGenerator(gen);

	exit (0);
}
